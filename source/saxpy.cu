﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include <time.h>
#include <math.h>

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define CLAMP(a, min, max) MIN((MAX((a), (min))),(max))
#define MEMORY_ASSIGNMENT_FEEDBACK_FRACTION 0.1

#define CUDA_CHECK(call)                                                              \
    do {                                                                              \
        hipError_t error = call;                                                     \
        if (error != hipSuccess) {                                                   \
            fprintf(stderr, "********* CUDA Error: %s, File: %s, Line: %d *********n",\
                    hipGetErrorString(error), __FILE__, __LINE__);                   \
            exit(1);                                                                  \
        }                                                                             \
    } while (0)

#define CHECK_NOT_NEG(call) if(call < 0){fprintf(stderr, "*********Error: %llu, File: %s, Line: %d *********n",call, __FILE__, __LINE__);exit(1);}
#define CALC_TIME(start_time, end_time) ((double)((end_time) - (start_time)) / CLOCKS_PER_SEC)

#ifdef SHORT
    #define ARRAY_VARIABLE_TYPE short
#elif INT
    #define ARRAY_VARIABLE_TYPE int
#elif LONG
    #define ARRAY_VARIABLE_TYPE long
#elif LONGLONG
    #define ARRAY_VARIABLE_TYPE long long
#elif FLOAT
    #define ARRAY_VARIABLE_TYPE float
#elif DOUBLE
    #define ARRAY_VARIABLE_TYPE double
#elif LONGDOUBLE
    #define ARRAY_VARIABLE_TYPE long double
#else
    #define ARRAY_VARIABLE_TYPE short
#endif


__global__ void SAXPY(ARRAY_VARIABLE_TYPE* d_a, ARRAY_VARIABLE_TYPE* d_b, float k, unsigned long long array_length)
{
    unsigned long long start_index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long stride = gridDim.x * blockDim.x;

    for(unsigned long long i = start_index; i < array_length; i+=stride)
        d_a[i] += k * d_b[i];
    return;
}

int FPUsPerSM(hipDeviceProp_t device_properties)
{
    int major = device_properties.major;
    int minor = device_properties.minor;
    switch (major) {
    case 2: // Fermi
        if (minor == 1) return 48;
        else return 32;
    case 3: // Kepler
        return 192;
    case 5: // Maxwell
        return 128;
    case 6: // Pascal
        if ((minor == 1) || (minor == 2)) return 128;
        else if (minor == 0) return 64;
    case 7: // Volta and Turing
        if ((minor == 0) || (minor == 5)) return 64;
    case 8: // Ampere
        if (minor == 0) return 64;
        else if (minor == 6) return 128;
        else if (minor == 9) return 128; // ada lovelace
    case 9: // Hopper
        if (minor == 0) return 128;
    }
    return NULL;
}

struct program_run_infomation
{
    double mem_usage_fraction;
    unsigned int profile;
    unsigned int oversubscription;
};

struct program_run_infomation default_program_run_information()
{
    struct program_run_infomation default_run_info = {0.9, 0, 0};
    return default_run_info;
}

void process_input_flag(char flag, char* assignment, struct program_run_infomation* program_info)
{
    switch(flag)
    {
        case 'm':
            program_info->mem_usage_fraction = atof(assignment);
            return;
        case 'p':
            program_info->profile = MAX(atoi(assignment), 0);
            return;
        case 's':
            program_info->oversubscription = MAX(atoi(assignment), 0);
            return;
    }
    printf("flag '%s' is not a valid flag. try running with the -h flag for a list of VALID flags.",assignment);exit(1);
}

//argv can contain the following;
//  memuseagefration -m : 0.0 - 1.0
//  profile -p : (0-UNINTMAX) --- should run through p times and calclate sd dev mean and such
//  oversubscription -s: (0-INTMAX)
int main(int argc, char* argv[])
{
    struct program_run_infomation run_info = default_program_run_information();

    for(int i = 1; i < argc; i++)
    {
        if(argv[i][0] == '-'){
            if(argv[i][1] != 'h'){process_input_flag(argv[i][1], argv[i+1], &run_info);i++;}
            else {printf("\nValid Flags:\n\t-a : arraytype {S, I, L, LL, F, D, LD}\n\t-m : memusagefraction {0.0-1.0}\n\t-p : profile {0, 1, 2, ...}\n\t-s : oversubscription {0, 1, 2, ...}\n\n");exit(1);}
        }
    }

    size_t size_of_list_element_bytes = sizeof(ARRAY_VARIABLE_TYPE);

    clock_t cpu_mem_alloc_time_start, cpu_mem_alloc_time_end;
    clock_t cpu_data_set_time_start, cpu_data_set_time_end;
    clock_t gpu_mem_alloc_time_start, gpu_mem_alloc_time_end;
    clock_t host_to_device_mem_copy_time_start, host_to_device_mem_copy_time_end;
    clock_t kernel_run_time_start, kernel_run_time_end;
    clock_t device_to_host_mem_copy_time_start, device_to_host_mem_copy_time_end;
    clock_t device_mem_free_time_start, device_mem_free_time_end;
    clock_t data_validation_time_start, data_validation_time_end;
    clock_t host_mem_free_time_start, host_mem_free_time_end;


    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);

    long long global_mem_on_gpu_bytes = device_properties.totalGlobalMem;
    unsigned long long size_of_array_to_add = global_mem_on_gpu_bytes * run_info.mem_usage_fraction / size_of_list_element_bytes;

    int number_of_sms = device_properties.multiProcessorCount;
    int number_of_fpus_per_sm = FPUsPerSM(device_properties);

    int max_threads_per_sm = device_properties.maxThreadsPerMultiProcessor;
    int max_threads_per_block = device_properties.maxThreadsPerBlock;
    int number_of_blocks = number_of_sms * max_threads_per_sm / max_threads_per_block;

    int number_of_threads_requested = (1 + run_info.oversubscription) * number_of_fpus_per_sm;
    int number_of_threads_per_block = MIN(max_threads_per_block, number_of_threads_requested);

    unsigned long long MEMORY_FEEDBACK_INDEX_SPLIT = floor(size_of_array_to_add * MEMORY_ASSIGNMENT_FEEDBACK_FRACTION);

    #ifndef OPTIMIZATION_O3
    printf("Allocating Host Memory\n");
    #endif

    //Asign variable
    if(run_info.profile > 0){cpu_mem_alloc_time_start = clock();}
    ARRAY_VARIABLE_TYPE* a = (ARRAY_VARIABLE_TYPE*)malloc(size_of_list_element_bytes * size_of_array_to_add);
    ARRAY_VARIABLE_TYPE* b = (ARRAY_VARIABLE_TYPE*)malloc(size_of_list_element_bytes * size_of_array_to_add);
    ARRAY_VARIABLE_TYPE* c = (ARRAY_VARIABLE_TYPE*)malloc(size_of_list_element_bytes * size_of_array_to_add);
    int k = 2;

    if (a == NULL || b == NULL || c == NULL){printf("NULL POINTER\na : %p\nb : %p\nc : %p", a, b, c);return -1;}

    #ifndef OPTIMIZATION_O3
    printf("Assigning Host Memory\n\n");
    #endif

    //set host data
    if(run_info.profile > 0){cpu_data_set_time_start = clock();}
    for (unsigned long long i = 0; i < size_of_array_to_add; i++)
    {

        #ifndef OPTIMIZATION_O3
        if (i % MEMORY_FEEDBACK_INDEX_SPLIT == 0)
        {
            printf("%lf %% complete\n", 100 * i / (double)size_of_array_to_add);
        }
        #endif
        a[i] = i;
        b[i] = i;
    }
    if(run_info.profile > 0){cpu_data_set_time_end = clock();}

    #ifndef OPTIMIZATION_O3
    printf("\nArray Size : %1.4lf * 10^9\nBlocks : %i\nThreads Per Block : %i\n\n",size_of_array_to_add / (double)1000000000, number_of_blocks, number_of_threads_per_block);
    #endif

    //define device pointers
    ARRAY_VARIABLE_TYPE* d_a;
    ARRAY_VARIABLE_TYPE* d_b;

    #ifndef OPTIMIZATION_O3
    printf("allocating device Memory\n");
    #endif

    //allocate device memory
    if(run_info.profile > 0){gpu_mem_alloc_time_start = clock();}
    CUDA_CHECK(hipMalloc(&d_a, size_of_list_element_bytes * size_of_array_to_add));
    CUDA_CHECK(hipMalloc(&d_b, size_of_list_element_bytes * size_of_array_to_add));
    if(run_info.profile > 0){gpu_mem_alloc_time_end = clock();}

    #ifndef OPTIMIZATION_O3
    printf("copying %lf GB from Host to Device\n", size_of_list_element_bytes * 2 * size_of_array_to_add / double(1024 * 1024 * 1024));
    #endif

    //cpy hist data to device
    if(run_info.profile > 0){host_to_device_mem_copy_time_start = clock();}
    CUDA_CHECK(hipMemcpy(d_a, a, size_of_list_element_bytes * size_of_array_to_add, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, size_of_list_element_bytes * size_of_array_to_add, hipMemcpyHostToDevice));
    if(run_info.profile > 0){host_to_device_mem_copy_time_end = clock();}

    #ifndef OPTIMIZATION_O3
    printf("Launching Kernel\n");
    #endif
    
    //launch kernel
    if(run_info.profile > 0){kernel_run_time_start = clock();}
    SAXPY<<<number_of_blocks, number_of_threads_per_block >>>(d_a, d_b, k, size_of_array_to_add);

    //not strictly needed as 'cudamemcpy' runs on the default stream as does 'Kernel' and hence it waits by default however if another stream was used, it would be mandatory
    CUDA_CHECK(hipDeviceSynchronize());
    if(run_info.profile > 0){kernel_run_time_end = clock();}
    
    #ifndef OPTIMIZATION_O3
    printf("Kernel Complete\n\n");
    printf("copying %lf GB from Device to Host\n", size_of_list_element_bytes * size_of_array_to_add / double(1024 * 1024 * 1024));
    #endif

    //read back data
    if(run_info.profile > 0){device_to_host_mem_copy_time_start = clock();}
    CUDA_CHECK(hipMemcpy(c, d_a, size_of_list_element_bytes * size_of_array_to_add, hipMemcpyDeviceToHost));
    if(run_info.profile > 0){device_to_host_mem_copy_time_end = clock();}

    #ifndef OPTIMIZATION_O3
    printf("Freeing Data from Device\n\n");
    #endif

    if(run_info.profile > 0){device_mem_free_time_start = clock();}
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipDeviceReset());
    if(run_info.profile > 0){device_mem_free_time_end = clock();}
    
    //VALIDATE RESULT
    if(run_info.profile > 0){data_validation_time_start = clock();}
    #ifndef OPTIMIZATION_O3
    printf("VALIDATING RESULT\n");
    #endif
    for (unsigned long long i = 0; i < size_of_array_to_add; i++)
    {
        if (c[i] != (ARRAY_VARIABLE_TYPE)((k + 1) * i))
        {
            //can't do this since format strings suck. if needed, make a function to return a format string given a var type
            printf("%i != %i\n", c[i], (ARRAY_VARIABLE_TYPE)((k + 1) * i));
            printf("RESULT INVALID\n\n");
            return -1;
        }
    }
    if(run_info.profile > 0){data_validation_time_end = clock();}
    //END VALIDATE RESULT

    #ifndef OPTIMIZATION_O3
    printf("RESULT VALID\n\n");
    printf("Freeing Data from Host\n\n");
    #endif

    if(run_info.profile > 0){host_mem_free_time_start = clock();} 
    free(a);
    free(b);
    free(c);
    if(run_info.profile > 0){host_mem_free_time_end = clock();} 

    if(run_info.profile > 0)
    {
        int number_of_active_threads_per_sm = MIN(number_of_fpus_per_sm, number_of_threads_requested);
        double percentage_of_fpus_used = 100 * number_of_active_threads_per_sm / (double)number_of_fpus_per_sm;
        double percentage_of_inactive_threads_used = 100 * (number_of_threads_per_block - number_of_active_threads_per_sm) / (double)(max_threads_per_block - number_of_active_threads_per_sm);
        
        printf("Number of Active Threads per SM : %i\nNumber of Active and Inactive Threads per SM : %i\nPercentage of FPUs used : %lf%%\nPercentage of Inactive Threads Used : %lf%%\nActive to Inactive Thread Ratio : (%i:%i)\n", number_of_active_threads_per_sm, number_of_threads_per_block, percentage_of_fpus_used, percentage_of_inactive_threads_used, number_of_active_threads_per_sm, (number_of_threads_per_block - number_of_active_threads_per_sm));
        printf("\n----------TIMINGS----------\n\n");
        printf("");
    }
    return 0;
}