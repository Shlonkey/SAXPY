﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>

#include <time.h>

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define CLAMP(a, min, max) MIN((MAX((a), (min))),(max))

#define CUDA_CHECK(call)                                                              \
    do {                                                                              \
        hipError_t error = call;                                                     \
        if (error != hipSuccess) {                                                   \
            fprintf(stderr, "********* CUDA Error: %s, File: %s, Line: %d *********n",\
                    hipGetErrorString(error), __FILE__, __LINE__);                   \
            exit(1);                                                                  \
        }                                                                             \
    } while (0)

#define CALC_TIME(start_time, end_time) ((double)((end_time) - (start_time)) / CLOCKS_PER_SEC)

__global__ void SAXPY(long long* d_a, long long* d_b, int k, long long array_length)
{
    long long start_index = blockIdx.x * blockDim.x + threadIdx.x;
    long long stride = gridDim.x * blockDim.x;

    for(long long i = start_index; i < array_length; i+=stride)
        d_a[i] += k * d_b[i];
    return;
}

int FPUsPerSM(hipDeviceProp_t device_properties)
{
    int major = device_properties.major;
    int minor = device_properties.minor;
    switch (major) {
    case 2: // Fermi
        if (minor == 1) return 48;
        else return 32;
    case 3: // Kepler
        return 192;
    case 5: // Maxwell
        return 128;
    case 6: // Pascal
        if ((minor == 1) || (minor == 2)) return 128;
        else if (minor == 0) return 64;
    case 7: // Volta and Turing
        if ((minor == 0) || (minor == 5)) return 64;
    case 8: // Ampere
        if (minor == 0) return 64;
        else if (minor == 6) return 128;
        else if (minor == 9) return 128; // ada lovelace
    case 9: // Hopper
        if (minor == 0) return 128;
    }
    return NULL;
}

enum arraytype
{
    SHORT,
    INT,
    LONG,
    LONGLONG,
    FLOAT,
    DOUBLE,
    LONGDOUBLE
};

struct program_run_infomation
{
    enum arraytype type_of_array;
    double mem_usage_fraction;
    unsigned int profile;
    unsigned int oversubscription;
};

struct program_run_infomation default_program_run_information()
{
    struct program_run_infomation default_run_info = {INT, 0.9, 0, 0};
    return default_run_info;
}

void process_input_flag(char flag, char* assignment, struct program_run_infomation* program_info)
{
    switch(flag)
    {
        case 'a':
            if(strcmp(assignment, "S") == 0){program_info->type_of_array=SHORT;}
            else if(strcmp(assignment, "I") == 0){program_info->type_of_array=INT;}
            else if(strcmp(assignment, "L") == 0){program_info->type_of_array=LONG;}
            else if(strcmp(assignment, "LL") == 0){program_info->type_of_array=LONGLONG;}
            else if(strcmp(assignment, "F") == 0){program_info->type_of_array=FLOAT;}
            else if(strcmp(assignment, "D") == 0){program_info->type_of_array=DOUBLE;}
            else if(strcmp(assignment, "LD") == 0){program_info->type_of_array=LONGDOUBLE;}
            break;
        case 'm':
            program_info->mem_usage_fraction = atof(assignment);
            break;
        case 'p':
            program_info->profile = MAX(atoi(assignment), 0);
            break;
        case 's':
            program_info->oversubscription = MAX(atoi(assignment), 0);
            break;
    }

}

size_t arrayTypeToBytes(enum arraytype type)
{
    switch(type)
    {
        case 0:
            return sizeof(short);
        case 1:
            return sizeof(int);
        case 2:
            return sizeof(long);
        case 3:
            return sizeof(long long);
        case 4:
            return sizeof(float);
        case 5:
            return sizeof(double);
        case 6:
            return sizeof(long double);
    }
    fprintf(stderr, "*********Error: %i, File: %s, Line: %d *********n",-1, __FILE__, __LINE__);
    exit(-1);
}

//argv can contain the following;
//  arraytype -a : {S, I, L, LL, F, D, LD}
//  memuseagefration -m : 0.0 - 1.0
//  profile -p : (0-UNINTMAX) --- should run through p times and calclate sd dev mean and such
//  oversubscription -s: (0-INTMAX)
int main(int argc, char* argv[])
{

    struct program_run_infomation run_info = default_program_run_information();

    for(int i = 1; i < argc; i++)
    {
        if(argv[i][0] == '-'){
            process_input_flag(argv[i][1], argv[i+1], &run_info);
            i++;
        }
    }

    size_t size_of_list_element_bytes = arrayTypeToBytes(run_info.type_of_array); 

    hipDeviceProp_t device_properties;
    hipGetDeviceProperties(&device_properties, 0);

    long long global_mem_on_gpu_bytes = device_properties.totalGlobalMem;
    long long size_of_array_to_add = global_mem_on_gpu_bytes * run_info.mem_usage_fraction / size_of_list_element_bytes;

    printf("%lli", size_of_array_to_add);

    int number_of_sms = device_properties.multiProcessorCount;
    int number_of_fpus_per_sm = FPUsPerSM(device_properties);

    int max_threads_per_sm = device_properties.maxThreadsPerMultiProcessor;
    int max_threads_per_block = device_properties.maxThreadsPerBlock;
    int number_of_blocks = number_of_sms * max_threads_per_sm / max_threads_per_block;

    int number_of_threads_requested = (1 + run_info.oversubscription) * number_of_fpus_per_sm;
    int number_of_threads_per_block = MIN(max_threads_per_block, number_of_threads_requested);

    #ifndef OPTIMIZATION_O3
    printf("Allocating Host Memory\n");
    #endif

    //Asign variable
    long long* a = (long long*)malloc(sizeof(long long) * size_of_array_to_add);
    long long* b = (long long*)malloc(sizeof(long long) * size_of_array_to_add);
    long long* c = (long long*)malloc(sizeof(long long) * size_of_array_to_add);
    int k = 3;

    if (a == NULL || b == NULL || c == NULL){printf("NULL POINTER\na : %p\nb : %p\nc : %p", a, b, c);return -1;}

    #ifndef OPTIMIZATION_O3
    printf("Assigning Host Memory\n\n");
    #endif
    if(run_info.type_of_array >= 0 && run_info.type_of_array <= 3)//integer type
    {
        for (unsigned long long i = 0; i < size_of_array_to_add; i++)
        {
            #ifndef OPTIMIZATION_O3
            if (i % 50000000 == 0)
            {
                printf("%lf %% complete\n", 100 * i / (double)size_of_array_to_add);
            }
            #endif
            a[i] = i;
            b[i] = i;
        }
    }
    else if (run_info.type_of_array >= 4 && run_info.type_of_array <= 6)//float type
    {
        for (unsigned long long i = 0; i < size_of_array_to_add; i++)
        {
            #ifndef OPTIMIZATION_O3
            if (i % 50000000 == 0)
            {
                printf("%lf %% complete\n", 100 * i / (double)size_of_array_to_add);
            }
            #endif
            a[i] = i / 2.0;
            b[i] = i / 2.0;
        }
    }

    #ifndef OPTIMIZATION_O3
    printf("\nArray Size : %1.4lf * 10^9\nBlocks : %i\nThreads Per Block : %i\n\n",size_of_array_to_add / (double)1000000000, number_of_blocks, number_of_threads_per_block);
    #endif

    //define device pointers
    long long* d_a;
    long long* d_b;

    #ifndef OPTIMIZATION_O3
    printf("allocating device Memory\n");
    #endif

    //allocate device memory
    CUDA_CHECK(hipMalloc(&d_a, sizeof(long long) * size_of_array_to_add));
    CUDA_CHECK(hipMalloc(&d_b, sizeof(long long) * size_of_array_to_add));

    #ifndef OPTIMIZATION_O3
    printf("copying %lf GB from Host to Device\n", sizeof(long long) * 2 * size_of_array_to_add / double(1024 * 1024 * 1024));
    #endif

    //cpy hist data to device
    CUDA_CHECK(hipMemcpy(d_a, a, sizeof(long long) * size_of_array_to_add, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b, sizeof(long long) * size_of_array_to_add, hipMemcpyHostToDevice));

    #ifndef OPTIMIZATION_O3
    printf("Launching Kernel\n");
    #endif
    
    //launch kernel
    SAXPY<<<number_of_blocks, number_of_threads_per_block >>>(d_a, d_b, k, size_of_array_to_add);

    //not strictly needed as 'cudamemcpy' runs on the default stream as does 'Kernel' and hence it waits by default however if another stream was used, it would be mandatory
    CUDA_CHECK(hipDeviceSynchronize());
    
    #ifndef OPTIMIZATION_O3
    printf("Kernel Complete\n\n");
    printf("copying %lf GB from Device to Host\n", sizeof(long long) * size_of_array_to_add / double(1024 * 1024 * 1024));
    #endif

    //read back data
    CUDA_CHECK(hipMemcpy(c, d_a, sizeof(long long) * size_of_array_to_add, hipMemcpyDeviceToHost));

    #ifndef OPTIMIZATION_O3
    printf("Freeing Data from Device\n\n");
    #endif

    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipDeviceReset());

    #ifndef OPTIMIZATION_O3
    printf("VALIDATING RESULT\n");
    if(run_info.type_of_array >= 0 && run_info.type_of_array <= 3)//integer type
    {
        for (long long i = 0; i < size_of_array_to_add; i++)
        {
            if (c[i] != (long long)(k + 1) * i)
            {
                printf("%lli != %lli\n", c[i], (long long)(k + 1) * i);
                printf("RESULT INVALID\n\n");
                return -1;
            }
        }
    }
    else if (run_info.type_of_array >= 4 && run_info.type_of_array <= 6)//float type
    {
        for (long long i = 0; i < size_of_array_to_add; i++)
        {
            if (c[i] != (long long)(k + 1) * i)
            {
                printf("%lli != %lli\n", c[i], (long long)(k + 1) * i / 2);
                printf("RESULT INVALID\n\n");
                return -1;
            }
        }
    }

    printf("RESULT VALID\n\n");
    printf("Freeing Data from Host\n\n");
    #endif

    free(a);
    free(b);
    free(c);

    if(run_info.profile > 0)
    {
        //print run metrics
        printf("");
        int number_of_active_threads_per_sm = MIN(number_of_fpus_per_sm, number_of_threads_requested);
        double percentage_of_fpus_used = 100 * number_of_active_threads_per_sm / (double)number_of_fpus_per_sm;
        double percentage_of_inactive_threads_used = 100 * (number_of_threads_per_block - number_of_active_threads_per_sm) / (double)(max_threads_per_block - number_of_active_threads_per_sm);
        
        printf("Number of Active Threads per SM : %i\nNumber of Active and Inactive Threads per SM : %i\nPercentage of FPUs used : %lf%%\nPercentage of Inactive Threads Used : %lf%%\nActive to Inactive Thread Ratio : (%i:%i)\n--------------------------------------------------------------------------------\n", number_of_active_threads_per_sm, number_of_threads_per_block, percentage_of_fpus_used, percentage_of_inactive_threads_used, number_of_active_threads_per_sm, (number_of_threads_per_block - number_of_active_threads_per_sm));
    }
    return 0;
}